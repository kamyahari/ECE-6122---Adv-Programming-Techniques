#include "hip/hip_runtime.h"
/*
Author: Kamya Hari
Class: ECE6122 A
Last Date Modified: 11/08/2024
Description:
Cuda kernel function to update the grids in Game of Life
*/

#include "cuda_kernels.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

// Static variables for device memory management
static bool* d_current = nullptr;
static bool* d_next = nullptr;
static size_t last_size = 0;

__global__ void updateGameOfLifeKernel(const bool* currentGrid, bool* nextGrid, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= width * height) return;

    int x = index % width;   // Column
    int y = index / width;   // Row
    int count = 0;

    // Count live neighbors
    for (int i = -1; i <= 1; ++i) {
        for (int j = -1; j <= 1; ++j) {
            if (i == 0 && j == 0) continue;

            int nx = x + i;
            int ny = y + j;

            // Check if neighbor indices are within bounds
            if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                count += currentGrid[ny * width + nx];
            }
        }
    }

    // Apply Game of Life rules
    nextGrid[index] = (currentGrid[index] && (count == 2 || count == 3)) || 
                     (!currentGrid[index] && count == 3);
}

void cleanupGameOfLife() {
    if (d_current != nullptr) {
        hipFree(d_current);
        hipFree(d_next);
        d_current = nullptr;
        d_next = nullptr;
        last_size = 0;
    }
}

void updateGameOfLife(bool* currentGrid, bool* nextGrid, int width, int height, int numThreads, MemoryType memoryType) {
    hipError_t err;
    size_t size = width * height * sizeof(bool);
    
    // Calculate grid dimensions
    int threadsPerBlock = numThreads;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;
    
    switch(memoryType) {
        case NORMAL: {
            // Regular CUDA memory allocation and transfers
            err = hipMalloc(&d_current, size);
            if (err != hipSuccess) {
                std::cerr << "NORMAL: Failed to allocate d_current: " << hipGetErrorString(err) << std::endl;
                return;
            }
            
            err = hipMalloc(&d_next, size);
            if (err != hipSuccess) {
                std::cerr << "NORMAL: Failed to allocate d_next: " << hipGetErrorString(err) << std::endl;
                hipFree(d_current);
                return;
            }
            
            err = hipMemcpy(d_current, currentGrid, size, hipMemcpyHostToDevice);
            if (err != hipSuccess) {
                std::cerr << "NORMAL: Failed to copy to device: " << hipGetErrorString(err) << std::endl;
                hipFree(d_current);
                hipFree(d_next);
                return;
            }
            
            updateGameOfLifeKernel<<<blocksPerGrid, threadsPerBlock>>>(d_current, d_next, width, height);
            hipDeviceSynchronize();
            
            err = hipMemcpy(nextGrid, d_next, size, hipMemcpyDeviceToHost);
            if (err != hipSuccess) {
                std::cerr << "NORMAL: Failed to copy from device: " << hipGetErrorString(err) << std::endl;
            }
            
            hipFree(d_current);
            hipFree(d_next);
            break;
        }
        
        case PINNED: {
            // Create pinned memory buffers for efficient transfers
            bool* h_pinned_current;
            bool* h_pinned_next;
            
            err = hipHostAlloc(&h_pinned_current, size, hipHostMallocDefault);
            if (err != hipSuccess) {
                std::cerr << "PINNED: Failed to allocate pinned current buffer: " << hipGetErrorString(err) << std::endl;
                return;
            }
            
            err = hipHostAlloc(&h_pinned_next, size, hipHostMallocDefault);
            if (err != hipSuccess) {
                std::cerr << "PINNED: Failed to allocate pinned next buffer: " << hipGetErrorString(err) << std::endl;
                hipHostFree(h_pinned_current);
                return;
            }
            
            // Copy input data to pinned memory
            memcpy(h_pinned_current, currentGrid, size);
            
            // Allocate device memory
            err = hipMalloc(&d_current, size);
            if (err != hipSuccess) {
                std::cerr << "PINNED: Failed to allocate d_current: " << hipGetErrorString(err) << std::endl;
                hipHostFree(h_pinned_current);
                hipHostFree(h_pinned_next);
                return;
            }
            
            err = hipMalloc(&d_next, size);
            if (err != hipSuccess) {
                std::cerr << "PINNED: Failed to allocate d_next: " << hipGetErrorString(err) << std::endl;
                hipFree(d_current);
                hipHostFree(h_pinned_current);
                hipHostFree(h_pinned_next);
                return;
            }
            
            // Transfer from pinned memory to device
            err = hipMemcpy(d_current, h_pinned_current, size, hipMemcpyHostToDevice);
            if (err != hipSuccess) {
                std::cerr << "PINNED: Failed to copy to device: " << hipGetErrorString(err) << std::endl;
                hipFree(d_current);
                hipFree(d_next);
                hipHostFree(h_pinned_current);
                hipHostFree(h_pinned_next);
                return;
            }
            
            updateGameOfLifeKernel<<<blocksPerGrid, threadsPerBlock>>>(d_current, d_next, width, height);
            hipDeviceSynchronize();
            
            // Transfer result to pinned memory
            err = hipMemcpy(h_pinned_next, d_next, size, hipMemcpyDeviceToHost);
            if (err != hipSuccess) {
                std::cerr << "PINNED: Failed to copy from device: " << hipGetErrorString(err) << std::endl;
            }
            
            // Copy result from pinned memory to output buffer
            memcpy(nextGrid, h_pinned_next, size);
            
            // Cleanup
            hipFree(d_current);
            hipFree(d_next);
            hipHostFree(h_pinned_current);
            hipHostFree(h_pinned_next);
            break;
        }
        
        case MANAGED: {
            // Use CUDA managed memory
            err = hipMallocManaged(&d_current, size);
            if (err != hipSuccess) {
                std::cerr << "MANAGED: Failed to allocate managed memory for d_current: " << hipGetErrorString(err) << std::endl;
                return;
            }
            
            err = hipMallocManaged(&d_next, size);
            if (err != hipSuccess) {
                std::cerr << "MANAGED: Failed to allocate managed memory for d_next: " << hipGetErrorString(err) << std::endl;
                hipFree(d_current);
                return;
            }
            
            // Copy initial state
            memcpy(d_current, currentGrid, size);
            
            updateGameOfLifeKernel<<<blocksPerGrid, threadsPerBlock>>>(d_current, d_next, width, height);
            err = hipDeviceSynchronize();
            if (err != hipSuccess) {
                std::cerr << "MANAGED: Kernel execution failed: " << hipGetErrorString(err) << std::endl;
            }
            
            // Copy result back
            memcpy(nextGrid, d_next, size);
            
            // Cleanup
            hipFree(d_current);
            hipFree(d_next);
            break;
        }
    }
    
    // Check for kernel errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
}